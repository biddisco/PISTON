#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>

#include "tetrarender.h"

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)

#define TETRA_BUFFER_SIZE 12000000
#define BASE_SCALE 3.5


TetraRender::TetraRender(char* a_filename, bool a_computeAverageIsovalue, float a_isovalue)
{
    strcpy(filename, a_filename);
    computeAverageIsovalue = a_computeAverageIsovalue;
    isovalue = a_isovalue;
    mouse_buttons = 0;
    translate = make_float3(0.0, 0.0, 0.0);
    wireMode = 0;
}


void TetraRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = 0.0 + zoomLevelBase*pct;
}


void TetraRender::resetView()
{
    qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    cameraFOV = 0.0 + zoomLevelBase*zoomLevelPct;
}


void TetraRender::display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    int minPass, maxPass;  minPass = maxPass = 0;
    if (wireMode == 0) { minPass = 0; maxPass = 1; }
    if (wireMode == 1) { minPass = 1; maxPass = 2; }
    if (wireMode == 2) { minPass = 0; maxPass = 2; }

isovalue *= 1.001;

    for (unsigned int pm=minPass; pm<maxPass; pm++)
    {
      isosurface->set_isovalue(isovalue);
      ((*isosurface)());

      if (!useInterop)
      {
        normals.assign(isosurface->normals_begin(), isosurface->normals_end());
        vertices.assign(isosurface->vertices_begin(), isosurface->vertices_end());
        colors.assign(thrust::make_transform_iterator(isosurface->scalars_begin(), color_map<float>(minValue, maxValue)),
    	            thrust::make_transform_iterator(isosurface->scalars_end(), color_map<float>(minValue, maxValue)));
      }
    
      glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);      
      glDisable(GL_POLYGON_OFFSET_LINE);
      if (pm == 1)
      {
        glEnable(GL_POLYGON_OFFSET_LINE);
        glPolygonOffset(1.0f,1.0f);
        glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
        glColor4f(1.0f, 1.0f, 1.0f, 1.0f);
      } 

      glMatrixMode(GL_PROJECTION);
      glLoadIdentity();
      gluPerspective(cameraFOV, 2.0f, BASE_SCALE/3.5f, 5.0f*BASE_SCALE); 

      glMatrixMode(GL_MODELVIEW);
      glLoadIdentity();
      gluLookAt(centerPos.x, -2.0f*BASE_SCALE, centerPos.z, centerPos.x, 0, centerPos.z, cameraUp.x, cameraUp.y, cameraUp.z); 
      glPushMatrix();

      glTranslatef(lookPos.x, lookPos.y, lookPos.z);
      if (pm == 1) glTranslatef(0.0f, -BASE_SCALE*0.03f, 0.0f);
      qrot.getRotMat(rotationMatrix);
      float3 offset = matrixMul(rotationMatrix, centerPos);

      glMultMatrixf(rotationMatrix);
      glTranslatef(offset.x-centerPos.x, offset.y-centerPos.y, offset.z-centerPos.z);

      glEnableClientState(GL_VERTEX_ARRAY);
      if (pm == 0) glEnableClientState(GL_COLOR_ARRAY);
      glEnableClientState(GL_NORMAL_ARRAY);

      #ifdef USE_INTEROP
        if (useInterop)
        {
          glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
          glVertexPointer(4, GL_FLOAT, 0, 0);
          glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
          glColorPointer(4, GL_FLOAT, 0, 0);
          glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
          glNormalPointer(GL_FLOAT, 0, 0);
          glDrawArrays(GL_TRIANGLES, 0, isosurface->num_total_vertices);
          glBindBuffer(GL_ARRAY_BUFFER, 0);
        }
        else
      #endif
      {
        if (showIso)
        {
          glNormalPointer(GL_FLOAT, 0, &normals[0]);
          glColorPointer(4, GL_FLOAT, 0, &colors[0]);
          glVertexPointer(4, GL_FLOAT, 0, &vertices[0]);
          glDrawArrays(GL_TRIANGLES, 0, vertices.size());
        }
      }

      glDisableClientState(GL_VERTEX_ARRAY);
      glDisableClientState(GL_COLOR_ARRAY);
      glDisableClientState(GL_NORMAL_ARRAY);

      glPopMatrix();
    }
}


void TetraRender::cleanup()
{
    #ifdef USE_INTEROP
      if (useInterop)
      {
        printf("Deleting VBO\n");
        if (vboBuffers[0])
        {
          for (int i=0; i<4; i++) hipGraphicsUnregisterResource(vboResources[i]);
	  for (int i=0; i<4; i++)
	  {
	    glBindBuffer(1, vboBuffers[i]);
	    glDeleteBuffers(1, &(vboBuffers[i]));
	    vboBuffers[i] = 0;
	  }
        }
      }
      else
    #endif
    {
      vertices.clear(); normals.clear(); colors.clear();
    }
}


void TetraRender::initGL(bool aAllowInterop)
{
    #ifdef USE_INTEROP
      useInterop = aAllowInterop;
    #else
      useInterop = false;
    #endif

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0, 0.0, 10.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    #ifdef USE_INTEROP
      if (useInterop)
      {
        glewInit();
        cudaGLSetGLDevice(0);

        glGenBuffers(4, vboBuffers);
        for (int i=0; i<3; i++)
        {
          unsigned int buffer_size = (i == 2) ? TETRA_BUFFER_SIZE*sizeof(float3) : TETRA_BUFFER_SIZE*sizeof(float4);
          glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[i]);
          glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
        }
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[3]);
        glBufferData(GL_ARRAY_BUFFER, TETRA_BUFFER_SIZE*sizeof(uint3), 0, GL_DYNAMIC_DRAW);

        glBindBuffer(GL_ARRAY_BUFFER, 0);
        for (int i=0; i<4; i++) hipGraphicsGLRegisterBuffer(&(vboResources[i]), vboBuffers[i], cudaGraphicsMapFlagsWriteDiscard);
      }
    #endif

    triFilter = vtkDataSetTriangleFilter::New();
    triFilter->TetrahedraOnlyOn();

    char fullFilename[1024];
    strcpy(fullFilename, filename);
    reader = vtkXMLUnstructuredGridReader::New();
    int fileFound = reader->CanReadFile(fullFilename);
    if (!fileFound)
    {
      sprintf(fullFilename, "%s/%s", STRINGIZE_VALUE_OF(DATA_DIRECTORY), filename);
      fileFound = reader->CanReadFile(fullFilename);
    }
    if (fileFound)
    {
      reader->SetFileName(fullFilename);  
      reader->Update();
      triFilter->SetInput(reader->GetOutput());
    } 
    else 
    {
      gridSize = atoi(filename);
      src = vtkRTAnalyticSource::New();
      src->SetWholeExtent(-gridSize, gridSize, -gridSize, gridSize, -gridSize, gridSize);
      src->Update();
      triFilter->SetInput(src->GetOutput());
    }

    triFilter->Update();
    tetrahedra = triFilter->GetOutput();

    double curVertex[3], minVertex[3], maxVertex[3];
    for (unsigned int i=0; i<3; i++) { minVertex[i] = DBL_MAX; maxVertex[i] = -DBL_MAX; }
    for (unsigned int i=0; i<tetrahedra->GetNumberOfPoints(); i++)
    {
      tetrahedra->GetPoint(i, curVertex);
      for (unsigned int j=0; j<3; j++)
      {
        if (minVertex[j] > curVertex[j]) minVertex[j] = curVertex[j];
        if (maxVertex[j] < curVertex[j]) maxVertex[j] = curVertex[j];
      }
    }
    double maxRange = 0.0;
    for (unsigned int i=0; i<3; i++) 
    { 
      double curRange = maxVertex[i] - minVertex[i];
      if (curRange > maxRange) maxRange = curRange; 
    }
    std::cout << "Range: " << maxRange << std::endl;
        
    utet = new unstructured_tetrahedra<SPACE>(tetrahedra, BASE_SCALE/maxRange);
    
    if (computeAverageIsovalue)
    {
      vtkDataArray* array1 = tetrahedra->GetPointData()->GetArray(0);
      vtkFloatArray* farray = vtkFloatArray::SafeDownCast(array1);
      float* rawData = farray->GetPointer(0);
      isovalue = 0.0f;
      for (unsigned int i=0; i<array1->GetNumberOfTuples(); i++) isovalue += rawData[i];
      isovalue /= (1.0f*array1->GetNumberOfTuples());
      std::cout << "Isovalue: " << isovalue << std::endl;
    }
    minValue = isovalue*0.9999f;  maxValue = isovalue;
 
    showIso = true;  
    zoomLevelBase = cameraFOV = 60.0f; cameraZ = 2.0f; zoomLevelPct = zoomLevelPctDefault = 0.5f;
    cameraFOV = zoomLevelBase*zoomLevelPct;  cameraUp = make_float3(0,0,1);
 
    isosurface = new marching_tetrahedron<unstructured_tetrahedra<SPACE>, unstructured_tetrahedra<SPACE> >(*utet, *utet, isovalue);

    ((*isosurface)());
    vertices.assign(isosurface->vertices_begin(), isosurface->vertices_end());
    lookPos = make_float3(0.0f, 0.0f, 0.0f);
    centerPos = make_float3(0.0f, 0.0f, 0.0f);
    for (unsigned int i=0; i<vertices.size(); i++) { centerPos.x += vertices[i].x;  centerPos.y += vertices[i].y;  centerPos.z += vertices[i].z; }
    centerPos.x /= vertices.size();  centerPos.y /= vertices.size();  centerPos.z /= vertices.size(); 
    printf("Center: %f %f %f\n", centerPos.x, centerPos.y, centerPos.z);

    isosurface->useInterop = useInterop;

    #ifdef USE_INTEROP
      if (useInterop)
      {
        for (int i=0; i<4; i++) isosurface->vboResources[i] = vboResources[i];
        isosurface->minIso = minValue;  isosurface->maxIso = maxValue;
      }
    #endif
}




