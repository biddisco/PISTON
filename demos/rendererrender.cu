#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>

#include "rendererrender.h"

#define TANGLE_EXAMPLE
//#define RTI_EXAMPLE
#define ORTHO

#define PACKED __attribute__((packed))

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)


struct Rect
{
    int left,top,right,bottom;
};

struct TGAHeader
{
    unsigned char  identsize		;   // size of ID field that follows 18 uint8 header (0 usually)
    unsigned char  colourmaptype	;   // type of colour map 0=none, 1=has palette
    unsigned char  imagetype		;   // type of image 0=none,1=indexed,2=rgb,3=grey,+8=rle packed

    unsigned short colourmapstart	PACKED;   // first colour map entry in palette
    unsigned short colourmaplength	PACKED;   // number of colours in palette
    unsigned char  colourmapbits	;         // number of bits per palette entry 15,16,24,32

    unsigned short xstart		PACKED;   // image x origin
    unsigned short ystart		PACKED;   // image y origin
    unsigned short width		PACKED;   // image width in pixels
    unsigned short height		PACKED;   // image height in pixels
    unsigned char  bits			;         // image bits per pixel 8,16,24,32
    unsigned char  descriptor		;         // image descriptor bits (vh flip bits)

    inline bool IsFlippedHorizontal() const
    {
      return (descriptor & 0x10) != 0;
    }

    inline bool IsFlippedVertical() const
    {
      return (descriptor & 0x20) != 0;
    }
};


RendererRender::RendererRender()
{
    mouse_buttons = 0;
    rcnt = 0;
    translate = make_float3(0.0, 0.0, 0.0);
    grid_size = 256;
    viewportWidth = 2*grid_size;  viewportHeight = 2*grid_size;
    //qDefault.set(1.0f, 1.0f, 0.0f, 1.0f); qDefault.normalize();
    qDefault.set(-0.27, -0.02, -0.71, 0.63); //0.0f, 0.0f, 0.0f, 1.0f);
    qDefault.normalize();
    resetView();
}


void RendererRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = 0.0 + zoomLevelBase*pct;
}


void RendererRender::resetView()
{
    qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    cameraFOV = 0.0 + zoomLevelBase*zoomLevelPct;
}


void RendererRender::display()
{
    struct timeval begin, end, diff;
    float seconds;
    gettimeofday(&begin, 0);

    //Quaternion newRotX;
    //newRotX.setEulerAngles(-0.2*50*3.14159/180.0, 0.0, 0.0);
    //qrot.mul(newRotX);
    qrot.getRotMat(rotationMatrix);
    float3 center;  center.x = center.y = center.z = 0.0f; //grid_size/2;
    float3 offset = make_float3(rotationMatrix[0]*center.x + rotationMatrix[1]*center.y + rotationMatrix[2]*center.z,
                                rotationMatrix[4]*center.x + rotationMatrix[5]*center.y + rotationMatrix[6]*center.z,
                                rotationMatrix[8]*center.x + rotationMatrix[9]*center.y + rotationMatrix[10]*center.z);
    offset.x = center.x - offset.x; offset.y = center.y - offset.y; offset.z = center.z - offset.z;

    isovalue += isoInc;
    if (isovalue > isoMax) { isovalue = isoMax; isoInc = -isoInc; }
    if (isovalue < isoMin) { isovalue = isoMin; isoInc = -isoInc; }
    printf("Isovalue: %f\n", isovalue);

#ifdef RTI_EXAMPLE
    (*isosurface2)();
    isosurface2->set_isovalue(isovalue);
    inputVertices.assign(isosurface2->vertices_begin(), isosurface2->vertices_end());
    inputNormals.assign(isosurface2->normals_begin(), isosurface2->normals_end());
    inputColors.assign(thrust::make_transform_iterator(isosurface2->scalars_begin(), color_map<float>(31.0f, 500.0f)),
                       thrust::make_transform_iterator(isosurface2->scalars_end(), color_map<float>(31.0f, 500.0f)));
#endif

#ifdef TANGLE_EXAMPLE
    (*isosurface)();
    inputVertices.assign(isosurface->vertices_begin(), isosurface->vertices_end());
    inputNormals.assign(isosurface->normals_begin(), isosurface->normals_end());
    inputColors.assign(thrust::make_transform_iterator(isosurface->scalars_begin(), color_map<float>(31.0f, 500.0f)),
                       thrust::make_transform_iterator(isosurface->scalars_end(), color_map<float>(31.0f, 500.0f)));
#endif

    renders->update(inputVertices.begin(), inputNormals.begin(), inputColors.begin(), inputVertices.size());

    inputVerticesHost = inputVertices;
    inputNormalsHost = inputNormals;
    inputColorsHost = inputColors;

#ifdef ORTHO
    //renders->setOrtho(0.0, grid_size, 0.0, grid_size, -2000.0f, 2000.0f);
    renders->setOrtho(-2.0f, 2.0f, -2.0f, 2.0f, -2000.0f, 2000.0f);
    renders->setRot(rotationMatrix);
    renders->translate(-offset.x, -offset.y, -offset.z);
#else
    renders->setPerspective(cameraFOV, viewportWidth/viewportHeight, 1.0f, 5.0f*grid_size);
    renders->setLookAt(make_float3(0,0,4.0f*grid_size), make_float3(0,0,0), make_float3(0,1,0));

    renders->rotate(rotationMatrix);
    renders->translate(-grid_size/2, -grid_size/2, -grid_size/2);
#endif
    renders->setLightProperties(make_float3(0.5f, 0.5f, 0.5f), make_float3(0.5f, 0.5f, 0.5f), 1.0f, 0.0f, 0.0f, make_float4(0.0f, 0.0f, 10000.0f, 1.0f));

    (*(renders))();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

#ifdef ORTHO
    //glOrtho(-0.0, grid_size, 0.0, grid_size, -2000.0f, 2000.0f);
    glOrtho(-2.0f, 2.0f, -2.0f, 2.0f, -2000.0f, 2000.0f);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMultMatrixf(rotationMatrix);
    glTranslatef(-offset.x, -offset.y, -offset.z);

    //glDisable(GL_LIGHTING);
#else
    gluPerspective(cameraFOV, viewportWidth/viewportHeight, 1.0f, 5.0f*grid_size);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0,0,4.0f*grid_size,0,0,0,0,1,0);

    glMultMatrixf(rotationMatrix);

    glTranslatef(-grid_size/2, -grid_size/2, -grid_size/2);
#endif

    glPushMatrix();

    glDisable(GL_CULL_FACE);

    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    glEnableClientState(GL_VERTEX_ARRAY);

    glNormalPointer(GL_FLOAT, 0, &inputNormalsHost[0]);
    glColorPointer(4, GL_FLOAT, 0, &inputColorsHost[0]);
    glVertexPointer(4, GL_FLOAT, 0, &inputVerticesHost[0]);
    glDrawArrays(GL_TRIANGLES, 0, inputVerticesHost.size());

    //if (rcnt == 0)
    //{
      //GLdouble mat1[16];
      //glGetDoublev(GL_PROJECTION_MATRIX,mat1);
      //GLdouble mat2[16];
      //glGetDoublev(GL_MODELVIEW_MATRIX,mat2);

      //std::cout << "OpenGL matrices" << std::endl;
      //for (unsigned int i=0; i<16; i++)  std::cout << mat1[(i%4)*4+(i/4)] << " ";  std::cout << std::endl;
      //for (unsigned int i=0; i<16; i++)  std::cout << mat2[(i%4)*4+(i/4)] << " ";  std::cout << std::endl;
    //}

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);

    glPopMatrix();

    char fname[128];  sprintf(fname, "test%d.tga", rcnt); //if (rcnt == 0)
    screenShot(fname, viewportWidth, viewportHeight, true);
    rcnt++;

    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "Total seconds: " << rcnt << " " << seconds << std::endl;

    //if (rcnt == 4) exit(-1);
}


void RendererRender::screenShot(std::string fileName, unsigned int width, unsigned int height, bool includeAlpha)
{
    std::cout << "Saving file" << std::endl;
    unsigned int pixelSize = 3;
    unsigned int pixelSizeBits = 24;
    GLenum pixelFormat = GL_BGR_EXT;

    if (includeAlpha)
    {
      pixelSize = sizeof(unsigned int);
      pixelSizeBits = 32;
      pixelFormat = GL_BGRA_EXT;
    }

    TGAHeader tgah;
    memset( &tgah,0,sizeof(TGAHeader) );

    tgah.bits = pixelSizeBits;
    tgah.height = height;
    tgah.width = width;
    tgah.imagetype = 2;

    std::ofstream ofile( fileName.c_str(), std::ios_base::binary );

    ofile.write( (char*)&tgah, sizeof(tgah) );
    thrust::host_vector<char> hostFrame;
    hostFrame.assign(renders->frame_begin(), renders->frame_end());
    ofile.write( &hostFrame[0], pixelSize*width*height );

    ofile.close();
}


void RendererRender::cleanup()
{

}


void RendererRender::initGL(bool aAllowInterop)
{
    glClearColor(1.0f, 1.0f, 1.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0, 0.0, 10000.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    //printf("Error code: %s\n", hipGetErrorString(errorCode));
    read();
}


int RendererRender::read()
{
#ifdef RTI_EXAMPLE
    reader = vtkXMLImageDataReader::New();
    char filename[1024];
    sprintf(filename, "%s/rti256.vti", STRINGIZE_VALUE_OF(DATA_DIRECTORY));
    int fileFound = reader->CanReadFile(filename);
    if (fileFound == 0) sprintf(filename, "rti256.vti");
    reader->SetFileName(filename);

    reader->Update();
    output = reader->GetOutput();
    image = new vtk_image3d<SPACE>(output);
    isovalue = 40.0f;  isoMax = 480.0f;  isoMin = 35.0f;  isoInc = 20.0f;
    isosurface2 = new marching_cube<vtk_image3d<SPACE>, vtk_image3d<SPACE> >(*image, *image, isovalue);
    isosurface2->useInterop = false;
    isosurface2->discardMinVals = true;
    (*isosurface2)();
#endif

#ifdef TANGLE_EXAMPLE
    tangle = new tangle_field<SPACE>(grid_size, grid_size, grid_size);
    isovalue = 0.2f;  isoMax = 0.9f;  isoMin = 0.1f;  isoInc = 0.1f;
    isosurface = new marching_cube<tangle_field<SPACE>,  tangle_field<SPACE> >(*tangle, *tangle, isovalue);
    (*isosurface)();
#endif

    /*inputVertices.push_back(make_float4(10.0f,  10.0f, 10.0f, 1.0f));  inputColors.push_back(make_float4(0.0f, 0.0f, 1.0f, 1.0f));
    inputVertices.push_back(make_float4(150.0f, 11.0f, 10.0f, 1.0f));  inputColors.push_back(make_float4(0.0f, 0.0f, 1.0f, 1.0f));
    inputVertices.push_back(make_float4(11.0f, 150.0f, 10.0f, 1.0f));  inputColors.push_back(make_float4(0.0f, 0.0f, 1.0f, 1.0f));

    inputVertices.push_back(make_float4(10.0f,  10.0f,  0.0f, 1.0f));  inputColors.push_back(make_float4(1.0f, 0.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(150.0f, 11.0f,  0.0f, 1.0f));  inputColors.push_back(make_float4(1.0f, 0.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(11.0f, 150.0f,  0.0f, 1.0f));  inputColors.push_back(make_float4(1.0f, 0.0f, 0.0f, 1.0f));*/

    inputVertices.push_back(make_float4(-3.0f, 3.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(-4.0f, 4.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(-5.0f, 2.0f, 0.0f, 1.0f));

    inputVertices.push_back(make_float4(4.0f, 0.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(5.0f, 4.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(3.0f, 3.0f, 0.0f, 1.0f));

    inputVertices.push_back(make_float4(-3.5f, 1.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(-4.0f, -1.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(-3.0f, -1.0f, 0.0f, 1.0f));

    inputVertices.push_back(make_float4(2.0f, -2.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(4.0f, -3.0f, 0.0f, 1.0f));
    inputVertices.push_back(make_float4(3.5f, -1.0f, 0.0f, 1.0f));

    for (unsigned int i=0; i<inputVertices.size(); i++) inputColors.push_back(make_float4(0.0f, 0.0f, 1.0f, 1.0f));
    for (unsigned int i=0; i<inputVertices.size(); i++) inputNormals.push_back(make_float3(0.0f, 0.0f, 1.0f));


#ifdef RTI_EXAMPLE
    inputVertices.assign(isosurface2->vertices_begin(), isosurface2->vertices_end());
    inputNormals.assign(isosurface2->normals_begin(), isosurface2->normals_end());
    inputColors.assign(thrust::make_transform_iterator(isosurface2->scalars_begin(), color_map<float>(31.0f, 500.0f)),
                       thrust::make_transform_iterator(isosurface2->scalars_end(), color_map<float>(31.0f, 500.0f)));
#endif

#ifdef TANGLE_EXAMPLE
    inputVertices.assign(isosurface->vertices_begin(), isosurface->vertices_end());
    inputNormals.assign(isosurface->normals_begin(), isosurface->normals_end());
    inputColors.assign(thrust::make_transform_iterator(isosurface->scalars_begin(), color_map<float>(31.0f, 500.0f)),
                       thrust::make_transform_iterator(isosurface->scalars_end(), color_map<float>(31.0f, 500.0f)));
#endif

    renders = new render<thrust::device_vector<float4>::iterator, thrust::device_vector<float3>::iterator, thrust::device_vector<float4>::iterator>(inputVertices.begin(),
                   inputNormals.begin(), inputColors.begin(), inputVertices.size(), viewportWidth, viewportHeight);

    zoomLevelBase = cameraFOV = 40.0; cameraZ = 2.0; zoomLevelPct = zoomLevelPctDefault = 0.5;
    center_pos = make_float3(0, 0, 0);
    cameraFOV = zoomLevelBase*zoomLevelPct;  camera_up = make_float3(0,1,0);

    inputVerticesHost = inputVertices;
    inputNormalsHost = inputNormals;
    inputColorsHost = inputColors;

    return 0;
}
